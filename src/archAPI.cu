/*
 * archAPI.cxx
 *
 *  Created on: Apr 10, 2018
 *      Author: snytav
 */

#include<stdlib.h>
#include<string.h>

#include "../include/archAPI.h"

#ifdef __HIPCC__
int MemoryCopy(void* dst,void *src,size_t size,int dir) {
   hipMemcpyKind cuda_dir;

   if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
   if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
   if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
   if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;

   return ((int)hipMemcpy(dst,src,size,cuda_dir));
}
#else

int MemoryCopy(void *dst, void *src, size_t size, int dir);

#endif

#ifdef __HIPCC__
int MemoryAllocate(void** dst,size_t size) {
   hipMalloc(dst,size);
   return 0;
}
#else

int MemoryAllocate(void **dst, size_t size);

#endif

#ifndef __HIPCC__

int getLastError() {
    return 0;
}

#else
int getLastError() {
    return (int)hipGetLastError();
}
#endif

#ifdef __HIPCC__
const char *getErrorString(int err) {
   return hipGetErrorString((hipError_t)err);
}
#else

const char *getErrorString(int err) { return ""; }

#endif

int get_num_args(void **args) {
    int i;
    for (i = 0; args[i] != NULL; i++);

    return i;
}

#ifndef __HIPCC__
dim3 threadIdx, blockIdx;
#endif