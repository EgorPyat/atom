/*
 * archAPI.cxx
 *
 *  Created on: Apr 10, 2018
 *      Author: snytav
 */

#include "../../include/archAPI.h"

#ifdef __HIPCC__
int MemoryCopy(void* dst,void *src,size_t size,int dir) {
    hipMemcpyKind cuda_dir;

    if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
    if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
    if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
    if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;

    int status = (int)hipMemcpy(dst,src,size,cuda_dir);

    return status != 0 ? -1 : 0;
}
#else

int MemoryCopy(void *dst, void *src, size_t size, int dir) {
    return 0;
}

#endif

#ifdef __HIPCC__
int MemoryAllocate(void** dst,size_t size) {
   int err = hipMalloc(dst, size);

   return err != 0 ? -1 : 0;
}
#else
int MemoryAllocate(void **dst, size_t size) {
    return 0;
}
#endif

#ifdef __HIPCC__
int getLastError() {
    return (int)hipGetLastError();
}
#else
int getLastError() {
    return 0;
}
#endif

#ifdef __HIPCC__
const char *getErrorString(int err) {
   return hipGetErrorString((hipError_t)err);
}
#else
const char *getErrorString(int err) {
    return "";
}
#endif
