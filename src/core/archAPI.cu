/*
 * archAPI.cxx
 *
 *  Created on: Apr 10, 2018
 *      Author: snytav
 */

#include "../../include/archAPI.h"

#ifdef __HIPCC__
int MemoryCopy(void* dst,void *src,size_t size,int dir) {
   hipMemcpyKind cuda_dir;

   if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
   if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
   if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
   if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;
// It's not correct! You can't return architecture specific result
// You must return Error or not_Error and in case of DEBUG  check the error here
   return ((int)hipMemcpy(dst,src,size,cuda_dir));
}
#else

int MemoryCopy(void *dst, void *src, size_t size, int dir);

#endif

#ifdef __HIPCC__
int MemoryAllocate(void** dst,size_t size) {
// ERROR: hipMalloc may failed. Check the error here and return it as it was describe in the comment above
   hipMalloc(dst,size);
   return 0;
}
#else

int MemoryAllocate(void **dst, size_t size);

#endif

// ERROR: why it's ihndef here while you have ifdef otherwice?
#ifndef __HIPCC__

int getLastError() {
    return 0;
}

#else
int getLastError() {
    return (int)hipGetLastError();
}
#endif

#ifdef __HIPCC__
const char *getErrorString(int err) {
   return hipGetErrorString((hipError_t)err);
}
#else

const char *getErrorString(int err) { return ""; }

#endif

#ifndef __HIPCC__
//dim3 threadIdx, blockIdx;
#endif
