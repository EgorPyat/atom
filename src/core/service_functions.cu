#include "hip/hip_runtime.h"
#include "../../include/service_functions.h"

using namespace std;

int setPrintfLimit() {
    size_t sizeP;

    std::cout << "Particle size " << sizeof(Particle) << " : " << sizeof(Particle) / sizeof(double) << ". CurrentTensor " << (int)sizeof(CurrentTensor) << " short " << (int)sizeof(char) << std::endl;

    hipDeviceGetLimit(&sizeP, hipLimitPrintfFifoSize);

    std::cout << "print default limit " << sizeP / 1024 / 1024 << std::endl;

    sizeP *= 10000;
    hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeP);

    hipDeviceGetLimit(&sizeP, hipLimitPrintfFifoSize);

    std::cout << "print limit set to " << sizeP / 1024 / 1024 << std::endl;

    return 0;
}

double CheckArraySilent(double *a, double *dbg_a, int size) {
    double diff = 0.0;

    for (int n = 0; n < size; n++) {
        diff += pow(a[n] - dbg_a[n], 2.0);
    }

    return pow(diff / (size), 0.5);
}

void hipMalloc3D(double **X, double **Y, double **Z, int nx, int ny, int nz) {
    hipMalloc((void **) X, sizeof(double) * (nx + 2) * (ny + 2) * (nz + 2));
    hipMalloc((void **) Y, sizeof(double) * (nx + 2) * (ny + 2) * (nz + 2));
    hipMalloc((void **) Z, sizeof(double) * (nx + 2) * (ny + 2) * (nz + 2));
}

void copyFieldsToGPU(
        double *d_Ex, double *d_Ey, double *d_Ez,
        double *d_Hx, double *d_Hy, double *d_Hz,
        double *d_Jx, double *d_Jy, double *d_Jz,
        double *d_npJx, double *d_npJy, double *d_npJz,
        double *d_Qx, double *d_Qy, double *d_Qz,
        double *Ex, double *Ey, double *Ez,
        double *Hx, double *Hy, double *Hz,
        double *Jx, double *Jy, double *Jz,
        double *npJx, double *npJy, double *npJz,
        double *Qx, double *Qy, double *Qz,
        int Nx, int Ny, int Nz
) {
    int err;

// TODO: It's better to define a macros and wrap  all MemoryCopy-s with it
// Moreover you dont need to write  numbers and any line... print __FILE__ and __LINE__ and optionaly
// You will print error code if build_type is debug (and not here but in ArchAPI.cu)

    err = MemoryCopy(d_Ex, Ex, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("1copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }
    err = MemoryCopy(d_Ey, Ey, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("2copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Ez, Ez, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("3copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Hx, Hx, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("4copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Hy, Hy, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("5copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Hz, Hz, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("6copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Jx, Jx, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("7copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Jy, Jy, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("8copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Jz, Jz, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("9copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_npJx, npJx, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("10copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_npJy, npJy, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("11copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_npJz, npJz, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("12copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Qx, Qx, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("13copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Qy, Qy, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("14copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }

    err = MemoryCopy(d_Qz, Qz, sizeof(double) * (Nx + 2) * (Ny + 2) * (Nz + 2), HOST_TO_DEVICE);
    if (err != hipSuccess) {
        printf("15copyFieldsToGPU err %d %s \n", err, getErrorString(err));
        exit(0);
    }
}

void InitGPUFields(
        double **d_Ex, double **d_Ey, double **d_Ez,
        double **d_Hx, double **d_Hy, double **d_Hz,
        double **d_Jx, double **d_Jy, double **d_Jz,
        double **d_npJx, double **d_npJy, double **d_npJz,
        double **d_Qx, double **d_Qy, double **d_Qz,
        double *Ex, double *Ey, double *Ez,
        double *Hx, double *Hy, double *Hz,
        double *Jx, double *Jy, double *Jz,
        double *npJx, double *npJy, double *npJz,
        double *Qx, double *Qy, double *Qz,
        int Nx, int Ny, int Nz
) {
    hipMalloc3D(d_Ex, d_Ey, d_Ez, Nx, Ny, Nz);
    hipMalloc3D(d_Hx, d_Hy, d_Hz, Nx, Ny, Nz);
    hipMalloc3D(d_Jx, d_Jy, d_Jz, Nx, Ny, Nz);
    hipMalloc3D(d_npJx, d_npJy, d_npJz, Nx, Ny, Nz);
    hipMalloc3D(d_Qx, d_Qy, d_Qz, Nx, Ny, Nz);

    copyFieldsToGPU(
            *d_Ex, *d_Ey, *d_Ez,
            *d_Hx, *d_Hy, *d_Hz,
            *d_Jx, *d_Jy, *d_Jz,
            *d_npJx, *d_npJy, *d_npJz,
            *d_Qx, *d_Qy, *d_Qz,
            Ex, Ey, Ez,
            Hx, Hy, Hz,
            Jx, Jy, Jz,
            npJx, npJy, npJz,
            Qx, Qy, Qz,
            Nx, Ny, Nz
    );
}


